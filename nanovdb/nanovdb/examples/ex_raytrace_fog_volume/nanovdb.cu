#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0

#include <cmath>
#include <chrono>

#include <hip/hip_runtime_api.h>

#include <nanovdb/util/IO.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include <nanovdb/util/Ray.h>
#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include "common.h"

#include <nanovdb/nanovtt/NanoVTT.h>
#include <nanovdb/nanovtt/util/GridHandle.h>
#include <nanovdb/nanovtt/util/SampleFromVoxels.h>

#if defined(NANOVDB_USE_CUDA)
using BufferT = nanovdb::CudaDeviceBuffer;
#else
using BufferT = nanovdb::HostBuffer;
#endif

namespace nanovdb {

template<typename RayT, typename AccT>
__hostdev__ inline bool firstActive(RayT& ray, AccT& acc, float& t)
{
    using namespace nanovdb;
    static const float Delta = 1.0001f;        // forward step-size along the ray to avoid getting stuck
    t = ray.t0();                              // initiate time
    Coord ijk = RoundDown<Coord>(ray.start()); // first voxel inside bbox
    for (HDDA<RayT, Coord> hdda(ray, acc.getDim(ijk, ray)); !acc.isActive(ijk); hdda.update(ray, acc.getDim(ijk, ray)))
    {
        if (!hdda.step())
            return false;                        // leap-frog HDDA and exit if ray bound is exceeded
        t = hdda.time() + Delta;                // update time
        ijk = RoundDown<Coord>(ray(t));       // update ijk
    }
    return true;
}

}

template<typename VoxelT>
void runNanoVDBInternal(nanovdb::GridHandle<BufferT>& handle, int numCPUIterations, int numGPUIterations, bool useHDDAAndSampler, std::string& imagePostFix, int width, int height, BufferT& imageBuffer)
{
    using GridT = nanovdb::NanoGrid<VoxelT>;
    using CoordT = nanovdb::Coord;
    using RealT = float;
    using Vec3T = nanovdb::Vec3<RealT>;
    using RayT = nanovdb::Ray<RealT>;

    auto* h_grid = handle.grid<VoxelT>();
    if (!h_grid)
        throw std::runtime_error("GridHandle does not contain a valid host grid");

    std::cout << "NanoVDB buffer size = " << static_cast<float>(handle.size()) / 1024 / 1024 << "MB" << std::endl;
    std::cout << "Number of grids = " << handle.gridCount() << std::endl;

    float* h_outImage = reinterpret_cast<float*>(imageBuffer.data());

    float              wBBoxDimZ = (float)h_grid->worldBBox().dim()[2] * 2;
    Vec3T              wBBoxCenter = Vec3T(h_grid->worldBBox().min() + h_grid->worldBBox().dim() * 0.5f);
    nanovdb::CoordBBox treeIndexBbox = h_grid->tree().bbox();
    std::cout << "Bounds: "
        << "[" << treeIndexBbox.min()[0] << "," << treeIndexBbox.min()[1] << "," << treeIndexBbox.min()[2] << "] -> ["
        << treeIndexBbox.max()[0] << "," << treeIndexBbox.max()[1] << "," << treeIndexBbox.max()[2] << "]" << std::endl;

    RayGenOp<Vec3T> rayGenOp(wBBoxDimZ, wBBoxCenter);
    CompositeOp     compositeOp;

    auto renderOpHDDA = [width, height, rayGenOp, compositeOp, treeIndexBbox] __hostdev__(int start, int end, float* image, const GridT * grid) {
        // get an accessor.
        auto acc = grid->tree().getAccessor();
        auto sampler = nanovdb::createSampler<1, decltype(acc), /*cache*/true>(acc);

        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // clip to bounds.
            if (iRay.clip(treeIndexBbox) == false) {
                compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                return;
#else
                continue;
#endif
            }
            float ti0 = iRay.t0(); // index-space hit params of the grid bbox
            float ti1 = iRay.t1();
            float ti01 = fabs(ti1 - ti0);
            {
                RayT fRay(iRay(ti0), iRay.dir(), 0.0, ti01);
                float tf;
                if (firstActive(fRay, acc, tf))
                    ti0 = ti0 + tf; // update ti0 to the first voxel hit
                else {
                    compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                    return;
#else
                    continue;
#endif
                }
            }
            {
                RayT bRay(iRay(ti1), -iRay.dir(), 0.0, ti01);
                float tb;
                if (firstActive(bRay, acc, tb))
                    ti1 = ti1 - tb; // update ti1 to the last voxel hit
                else {
                    compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                    return;
#else
                    continue;
#endif
                }
            }
            if (ti1 <= ti0) {
                compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                return;
#else
                continue;
#endif
            }
            ti0 -= 1.f; // expand by 1 voxel to ensure DDA march covers entirety of touched voxels
            ti1 += 1.f;

            // integrate...
            const float dt = 0.5f;
            float       transmittance = 1.0f;
            for (float t = ti0; t < ti1; t += dt) {
                float sigma = sampler(iRay(t)) * 0.1f;
                transmittance *= 1.0f - sigma * dt;
            }
            // write transmittance.
            compositeOp(image, i, width, height, 0.0f, 1.0f - transmittance);
        }
    };


    auto renderOp = [width, height, rayGenOp, compositeOp, treeIndexBbox] __hostdev__(int start, int end, float* image, const GridT* grid) {
        auto acc = grid->tree().getAccessor();

        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // clip to bounds.
            if (iRay.clip(treeIndexBbox) == false) {
                compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                return;
#else
                continue;
#endif
            }
            // integrate...
            const float dt = 0.5f;
            float       transmittance = 1.0f;
            for (float t = iRay.t0(); t < iRay.t1(); t += dt) {
                float sigma = acc.getValue(CoordT::Floor(iRay(t))) * 0.1f;
                transmittance *= 1.0f - sigma * dt;
            }
            // write transmittance.
            compositeOp(image, i, width, height, 0.0f, 1.0f - transmittance);
        }
    };

    if (numCPUIterations > 0)
    {
        float durationAvg = 0;
        for (int i = 0; i < numCPUIterations; ++i) 
        {
            float duration = 0;
            if (useHDDAAndSampler) {
                duration = renderImage(false, renderOpHDDA, width, height, h_outImage, h_grid);
            }
            else {
                duration = renderImage(false, renderOp, width, height, h_outImage, h_grid);
            }
            //std::cout << "Duration(NanoVDB-Host) = " << duration << " ms" << std::endl;
            durationAvg += duration;
        }
        durationAvg /= numCPUIterations;
        std::cout << "Average Duration(NanoVDB-Host) = " << durationAvg << " ms" << std::endl;
		std::string fileName = std::string("raytrace_fog_volume") + imagePostFix + "-nanovdb-host.pfm";
		saveImage(fileName, width, height, (float*)imageBuffer.data());
    }

#if defined(NANOVDB_USE_CUDA)
    if (numGPUIterations > 0) {
        handle.deviceUpload();

        auto* d_grid = handle.deviceGrid<VoxelT>();
        if (!d_grid)
            throw std::runtime_error("GridHandle does not contain a valid device grid");

        imageBuffer.deviceUpload();
        float* d_outImage = reinterpret_cast<float*>(imageBuffer.deviceData());

        {
            float durationAvg = 0;
            for (int i = 0; i < numGPUIterations; ++i)
            {
                float duration = 0;
                if (useHDDAAndSampler) {
                    duration = renderImage(true, renderOpHDDA, width, height, d_outImage, d_grid);
                }
                else {
                    duration = renderImage(true, renderOp, width, height, d_outImage, d_grid);
                }
                //std::cout << i << ": Duration(NanoVDB-Cuda) = " << duration << " ms" << std::endl;
                durationAvg += duration;
            }
            durationAvg /= numGPUIterations;
            std::cout << "Average Duration(NanoVDB-Cuda) = " << durationAvg << " ms" << std::endl;
            imageBuffer.deviceDownload();
			std::string fileName = std::string("raytrace_fog_volume") + imagePostFix + "-nanovdb-cuda.pfm";
			saveImage(fileName, width, height, (float*)imageBuffer.data());
        }
    }
#endif
}

void runNanoVDB(nanovdb::GridHandle<BufferT>& handle, int numCPUIterations, int numGPUIterations, bool useHDDAAndSampler, std::string& imagePostFix, int width, int height, BufferT& imageBuffer)
{
    if (handle.grid<float>()) {
        runNanoVDBInternal<float>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else if (handle.grid<nanovdb::Fp4>()) {
        runNanoVDBInternal<nanovdb::Fp4>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else if (handle.grid<nanovdb::Fp8>()) {
        runNanoVDBInternal<nanovdb::Fp8>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else if (handle.grid<nanovdb::FpN>()) {
        runNanoVDBInternal<nanovdb::FpN>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else
        throw std::runtime_error("GridHandle does not contain a valid host grid");
}


namespace nanovtt {

constexpr bool useCachedSampler = false;

template<typename RayT, typename AccT>
__hostdev__ inline bool firstActive(RayT& ray, AccT& acc, float& t)
{
    using namespace nanovdb;
    static const float Delta = 1.0001f;        // forward step-size along the ray to avoid getting stuck
    t = ray.t0();                              // initiate time
    Coord ijk = RoundDown<Coord>(ray.start()); // first voxel inside bbox
    for (HDDA<RayT, Coord> hdda(ray, acc.getDim(ijk, ray)); !acc.isActive(ijk); hdda.update(ray, acc.getDim()))
    {
        if (!hdda.step())
            return false;                        // leap-frog HDDA and exit if ray bound is exceeded
        t = hdda.time() + Delta;                // update time
        ijk = RoundDown<Coord>(ray(t));       // update ijk
    }
    return true;
}

}

template<typename VoxelT>
void runNanoVTTInternal(nanovtt::GridHandle<BufferT>& handle, int numCPUIterations, int numGPUIterations, bool useHDDAAndSampler, std::string& imagePostFix, int width, int height, BufferT& imageBuffer)
{
    using GridT = nanovtt::Grid<nanovtt::Tree<VoxelT>>;
    using CoordT = nanovdb::Coord;
    using RealT = float;
    using Vec3T = nanovdb::Vec3<RealT>;
    using RayT = nanovdb::Ray<RealT>;

    auto* h_grid = handle.grid<VoxelT>();
    if (!h_grid)
        throw std::runtime_error("GridHandle does not contain a valid host grid");

    std::cout << "NanoVTT buffer size = " << static_cast<float>(handle.size()) / 1024 / 1024 << "MB" << std::endl;
    std::cout << "Number of grids = " << handle.gridCount() << std::endl;

    float* h_outImage = reinterpret_cast<float*>(imageBuffer.data());

    float              wBBoxDimZ = (float)h_grid->worldBBox().dim()[2] * 2;
    Vec3T              wBBoxCenter = Vec3T(h_grid->worldBBox().min() + h_grid->worldBBox().dim() * 0.5f);
    nanovdb::CoordBBox treeIndexBbox = h_grid->tree().bbox();
    std::cout << "Bounds: "
        << "[" << treeIndexBbox.min()[0] << "," << treeIndexBbox.min()[1] << "," << treeIndexBbox.min()[2] << "] -> ["
        << treeIndexBbox.max()[0] << "," << treeIndexBbox.max()[1] << "," << treeIndexBbox.max()[2] << "]" << std::endl;

    RayGenOp<Vec3T> rayGenOp(wBBoxDimZ, wBBoxCenter);
    CompositeOp     compositeOp;

    auto renderOpHDDA = [width, height, rayGenOp, compositeOp, treeIndexBbox] __hostdev__(int start, int end, float* image, const GridT * grid) {
        // get an accessor.
        auto acc = grid->tree().getAccessor();
        auto sampler = nanovtt::createSampler<1, decltype(acc), /*cache*/nanovtt::useCachedSampler>(acc);

        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // clip to bounds.
            if (iRay.clip(treeIndexBbox) == false) {
                compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                return;
#else
                continue;
#endif
            }
            float ti0 = iRay.t0(); // index-space hit params of the grid bbox
            float ti1 = iRay.t1();
            float ti01 = fabs(ti1 - ti0);
            {
                RayT fRay(iRay(ti0), iRay.dir(), 0.0, ti01);
                float tf;
                if (nanovtt::firstActive(fRay, acc, tf))
                    ti0 = ti0 + tf; // update ti0 to the first voxel hit
                else {
                    compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                    return;
#else
                    continue;
#endif
                }
            }
            {
                RayT bRay(iRay(ti1), -iRay.dir(), 0.0, ti01);
                float tb;
                if (nanovtt::firstActive(bRay, acc, tb))
                    ti1 = ti1 - tb; // update ti1 to the last voxel hit
                else {
                    compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                    return;
#else
                    continue;
#endif
                }
            }
            if (ti1 <= ti0) {
                compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                return;
#else
                continue;
#endif
            }
            ti0 -= 1.f; // expand by 1 voxel to ensure DDA march covers entirety of touched voxels
            ti1 += 1.f;

            // integrate...
            const float dt = 0.5f;
            float       transmittance = 1.0f;
            for (float t = ti0; t < ti1; t += dt) {
                float sigma = sampler(iRay(t)) * 0.1f;
                transmittance *= 1.0f - sigma * dt;
            }
            // write transmittance.
            compositeOp(image, i, width, height, 0.0f, 1.0f - transmittance);
        }
    };


    auto renderOp = [width, height, rayGenOp, compositeOp, treeIndexBbox] __hostdev__(int start, int end, float* image, const GridT * grid) {
        auto acc = grid->tree().getAccessor();
        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // clip to bounds.
            if (iRay.clip(treeIndexBbox) == false) {
                compositeOp(image, i, width, height, 0.0f, 0.0f);
#ifdef __CUDA_ARCH__
                return;
#else
                continue;
#endif
            }
            // integrate...
            const float dt = 0.5f;
            float       transmittance = 1.0f;          
            for (float t = iRay.t0(); t < iRay.t1(); t += dt) {
                float sigma = acc.getValue(CoordT::Floor(iRay(t))) * 0.1f;
                transmittance *= 1.0f - sigma * dt;
        }
            // write transmittance.
            compositeOp(image, i, width, height, 0.0f, 1.0f - transmittance);
        }
    };

    if (numCPUIterations > 0)
    {
        float durationAvg = 0;
        for (int i = 0; i < numCPUIterations; ++i) 
        {
            float duration = 0;
            if (useHDDAAndSampler) {
                duration = renderImage(false, renderOpHDDA, width, height, h_outImage, h_grid);
            }
            else {
                duration = renderImage(false, renderOp, width, height, h_outImage, h_grid);
            }
            //std::cout << "Duration(NanoVTT-Host) = " << duration << " ms" << std::endl;
            durationAvg += duration;
        }
        durationAvg /= numCPUIterations;
        std::cout << "Average Duration(NanoVTT-Host) = " << durationAvg << " ms" << std::endl;
		std::string fileName = std::string("raytrace_fog_volume") + imagePostFix + "-nanovtt-host.pfm";
		saveImage(fileName, width, height, (float*)imageBuffer.data());
    }

#if defined(NANOVDB_USE_CUDA)
    if (numGPUIterations > 0) {
        handle.deviceUpload();

        auto* d_grid = handle.deviceGrid<VoxelT>();
        if (!d_grid)
            throw std::runtime_error("GridHandle does not contain a valid device grid");

        imageBuffer.deviceUpload();
        float* d_outImage = reinterpret_cast<float*>(imageBuffer.deviceData());

		float durationAvg = 0;
		for (int i = 0; i < numGPUIterations; ++i)
		{
			float duration = 0;
			if (useHDDAAndSampler) {
				duration = renderImage(true, renderOpHDDA, width, height, d_outImage, d_grid);
			}
			else {
				duration = renderImage(true, renderOp, width, height, d_outImage, d_grid);
			}
			//std::cout << i << ": Duration(NanoVTT-Cuda) = " << duration << " ms" << std::endl;
			durationAvg += duration;
		}
		durationAvg /= numGPUIterations;
		std::cout << "Average Duration(NanoVTT-Cuda) = " << durationAvg << " ms" << std::endl;
		imageBuffer.deviceDownload();
		std::string fileName = std::string("raytrace_fog_volume") + imagePostFix + "-nanovtt-cuda.pfm";
		saveImage(fileName, width, height, (float*)imageBuffer.data());
    }
#endif
}

void runNanoVTT(nanovtt::GridHandle<BufferT>& handle, int numCPUIterations, int numGPUIterations, bool useHDDAAndSampler, std::string& imagePostFix, int width, int height, BufferT& imageBuffer)
{
    if (handle.grid<float>()) {
        runNanoVTTInternal<float>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else if (handle.grid<nanovdb::Fp4>()) {
        runNanoVTTInternal<nanovdb::Fp4>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else if (handle.grid<nanovdb::Fp8>()) {
        runNanoVTTInternal<nanovdb::Fp8>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else if (handle.grid<nanovdb::FpN>()) {
        runNanoVTTInternal<nanovdb::FpN>(handle, numCPUIterations, numGPUIterations, useHDDAAndSampler, imagePostFix, width, height, imageBuffer);
    }
    else
        throw std::runtime_error("GridHandle does not contain a valid host grid");
}
