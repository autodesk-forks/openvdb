#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0

#include <cmath>
#include <chrono>

#include <nanovdb/util/IO.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include <nanovdb/util/Ray.h>
#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/GridBuilder.h>

#include "common.h"
#include <nanovdb/nanovtt/NanoVTT.h>
#include <nanovdb/nanovtt/util/GridHandle.h>
#include <nanovdb/nanovtt/util/HDDA.h>
#include <nanovdb/nanovtt/util/SampleFromVoxels.h>

#define PNANOVDB_C
#define PNANOVDB_HDDA
#include <nanovdb/nanovtt/PNanoVTT.h>

#if defined(NANOVDB_USE_CUDA)
using BufferT = nanovdb::CudaDeviceBuffer;
#else
using BufferT = nanovdb::HostBuffer;
#endif


void runNanoVDB(nanovdb::GridHandle<BufferT>& handle, int numCPUIterations, int numGPUIterations, std::string& imagePostFix, int width, int height, BufferT& imageBuffer)
{
    using GridT = nanovdb::FloatGrid;
    using CoordT = nanovdb::Coord;
    using RealT = float;
    using Vec3T = nanovdb::Vec3<RealT>;
    using RayT = nanovdb::Ray<RealT>;

    auto* h_grid = handle.grid<float>();
    if (!h_grid)
        throw std::runtime_error("GridHandle does not contain a valid host grid");

    std::cout << "NanoVDB buffer size = " << static_cast<float>(handle.size()) / 1024 / 1024 << "MB" << std::endl;
    std::cout << "Number of grids = " << handle.gridCount() << std::endl;

    float* h_outImage = reinterpret_cast<float*>(imageBuffer.data());

    float              wBBoxDimZ = (float)h_grid->worldBBox().dim()[2] * 2;
    Vec3T              wBBoxCenter = Vec3T(h_grid->worldBBox().min() + h_grid->worldBBox().dim() * 0.5f);
    nanovdb::CoordBBox treeIndexBbox = h_grid->tree().bbox();
    std::cout << "Bounds: "
              << "[" << treeIndexBbox.min()[0] << "," << treeIndexBbox.min()[1] << "," << treeIndexBbox.min()[2] << "] -> ["
              << treeIndexBbox.max()[0] << "," << treeIndexBbox.max()[1] << "," << treeIndexBbox.max()[2] << "]" << std::endl;

    RayGenOp<Vec3T> rayGenOp(wBBoxDimZ, wBBoxCenter);
    CompositeOp     compositeOp;

    auto renderOp = [width, height, rayGenOp, compositeOp, treeIndexBbox, wBBoxDimZ] __hostdev__(int start, int end, float* image, const GridT* grid) {
        // get an accessor.
        auto acc = grid->tree().getAccessor();

        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // intersect...
            float  t0;
            CoordT ijk;
            float  v;
            if (nanovdb::ZeroCrossing(iRay, acc, ijk, v, t0)) {
                // write distance to surface. (we assume it is a uniform voxel)
                float wT0 = t0 * float(grid->voxelSize()[0]);
                compositeOp(image, i, width, height, wT0 / (wBBoxDimZ * 2), 1.0f);
            } else {
                // write background value.
                compositeOp(image, i, width, height, 0.0f, 0.0f);
            }
        }
    };

    if (numCPUIterations > 0)
    {
        float durationAvg = 0;
        for (int i = 0; i < numCPUIterations; ++i) {
            float duration = renderImage(false, renderOp, width, height, h_outImage, h_grid);
            //std::cout << "Duration(NanoVDB-Host) = " << duration << " ms" << std::endl;
            durationAvg += duration;
        }
        durationAvg /= numCPUIterations;
        std::cout << "Average Duration(NanoVDB-Host) = " << durationAvg << " ms" << std::endl;

		saveImage(std::string("raytrace_level_set") + imagePostFix + "-nanovdb-host.pfm", width, height, (float*)imageBuffer.data());
    }

#if defined(NANOVDB_USE_CUDA)
    if (numGPUIterations > 0)
    {
        handle.deviceUpload();

        auto* d_grid = handle.deviceGrid<float>();
        if (!d_grid)
            throw std::runtime_error("GridHandle does not contain a valid device grid");

        imageBuffer.deviceUpload();
        float* d_outImage = reinterpret_cast<float*>(imageBuffer.deviceData());

        {
            float durationAvg = 0;
            for (int i = 0; i < numGPUIterations; ++i) {
                float duration = renderImage(true, renderOp, width, height, d_outImage, d_grid);
                //std::cout << "Duration(NanoVDB-Cuda) = " << duration << " ms" << std::endl;
                durationAvg += duration;
            }
            durationAvg /= numGPUIterations;
            std::cout << "Average Duration(NanoVDB-Cuda) = " << durationAvg << " ms" << std::endl;

            imageBuffer.deviceDownload();
            saveImage(std::string("raytrace_level_set") + imagePostFix + "-nanovdb-cuda.pfm", width, height, (float*)imageBuffer.data());
        }
    }
#endif
}


void runNanoVTT(nanovtt::GridHandle<BufferT>& handle, int numCPUIterations, int numGPUIterations, bool useSubVoxelAccuracy, std::string& imagePostFix, int width, int height, BufferT& imageBuffer)
{
    using GridT = nanovtt::FloatGrid;
    using CoordT = nanovdb::Coord;
    using RealT = float;
    using Vec3T = nanovdb::Vec3<RealT>;
    using RayT = nanovdb::Ray<RealT>;

    auto* h_grid = handle.template grid<float>();
    if (!h_grid)
        throw std::runtime_error("GridHandle does not contain a valid host grid");

    std::cout << "NanoVTT buffer size = " << static_cast<float>(handle.size()) / 1024 / 1024 << "MB" << std::endl;
    std::cout << "Number of grids = " << handle.gridCount() << std::endl;

    float* h_outImage = reinterpret_cast<float*>(imageBuffer.data());

    float              wBBoxDimZ = (float)h_grid->worldBBox().dim()[2] * 2;
    Vec3T              wBBoxCenter = Vec3T(h_grid->worldBBox().min() + h_grid->worldBBox().dim() * 0.5f);
    nanovdb::CoordBBox treeIndexBbox = h_grid->tree().bbox();
    std::cout << "Bounds: "
        << "[" << treeIndexBbox.min()[0] << "," << treeIndexBbox.min()[1] << "," << treeIndexBbox.min()[2] << "] -> ["
        << treeIndexBbox.max()[0] << "," << treeIndexBbox.max()[1] << "," << treeIndexBbox.max()[2] << "]" << std::endl;

    RayGenOp<Vec3T> rayGenOp(wBBoxDimZ, wBBoxCenter);
    CompositeOp     compositeOp;

    // HDDA with sampler and sub-voxel accuracy
    auto renderOpSubVoxel = [width, height, rayGenOp, compositeOp, treeIndexBbox, wBBoxDimZ] __hostdev__(int start, int end, float* image, const GridT * grid) {
        auto acc = grid->tree().getAccessor();
        auto sampler = nanovtt::createSampler<1>(acc);
        constexpr float isoValue = 0.f;
        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // intersect...
            float  t0 = 0.f;
            if (nanovtt::zeroCrossingSubVoxel(
                isoValue, iRay, sampler, t0)) {
                // write distance to surface. (we assume it is a uniform voxel)
                float wT0 = t0 * float(grid->voxelSize()[0]);
                compositeOp(image, i, width, height, wT0 / (wBBoxDimZ * 2), 1.0f);
            }
            else {
                // write background value.
                compositeOp(image, i, width, height, 0.0f, 0.0f);
            }
        }
    };

    // HDDA with sampler and sub-voxel accuracy for PNanoVTT
    auto renderOpSubVoxelPNanoVTT = [width, height, rayGenOp, compositeOp, treeIndexBbox, wBBoxDimZ] (int start, int end, float* image, const GridT * grid) {

        const pnanovdb_buf_t buf = pnanovdb_make_buf(const_cast<uint32_t*>(
            reinterpret_cast<uint32_t const*>(grid->tree().getContainer())),
            grid->tree().getContainer()->m_bufferSize);
        const pnanovdb_grid_handle_t pGridHandle{ pnanovdb_address_offset64(pnanovdb_address_null(), grid->tree().getContainer()->m_containerSize) };
        const pnanovtt_tree_handle_t pTreeHandle{ pnanovdb_address_offset64(pnanovdb_address_null(), grid->tree().getContainer()->m_containerSize + PNANOVDB_GRID_SIZE) };

        pnanovtt_cached_sampler_t psampler;
        pnanovtt_cached_sampler_init(buf, PNANOVDB_REF(psampler), pGridHandle, pTreeHandle);

        constexpr float isoValue = 0.f;
        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // intersect...
            float  t0 = 0.f;
            float vHit = 0.f;
            pnanovdb_vec3_t pHit;
            const pnanovdb_vec3_t origin{ iRay.eye()[0], iRay.eye()[1], iRay.eye()[2] };
            const pnanovdb_vec3_t direction{ iRay.dir()[0], iRay.dir()[1], iRay.dir()[2] };
            const float tMin = iRay.t0();
            const float tMax = iRay.t1();

            const pnanovdb_bool_t foundZC = pnanovtt_hdda_zero_crossing_sub_voxel(
                buf,
                PNANOVDB_REF(psampler),
                PNANOVDB_REF(origin),
                tMin,
                PNANOVDB_REF(direction),
                tMax,
                isoValue,
                PNANOVDB_REF(t0),
                PNANOVDB_REF(vHit),
                PNANOVDB_REF(pHit),
                PNANOVTT_TRAITS_MAX_DEPTH);
            if (foundZC) {
                // write distance to surface. (we assume it is a uniform voxel)
                float wT0 = t0 * float(grid->voxelSize()[0]);
                compositeOp(image, i, width, height, wT0 / (wBBoxDimZ * 2), 1.0f);
            }
            else {
                // write background value.
                compositeOp(image, i, width, height, 0.0f, 0.0f);
            }
        }
    };

    // Original HDDA approach
    auto renderOp = [width, height, rayGenOp, compositeOp, treeIndexBbox, wBBoxDimZ] __hostdev__(int start, int end, float* image, const GridT * grid) {
        auto acc = grid->tree().getAccessor();

        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // intersect...
            float  t0;
            CoordT ijk;
            float  v;
            // Note: A few changes were required to nanovdb::ZeroCrossing to make it work with adaptive volumes.
            //       Hence we use nanovtt::ZeroCrossing here.
            if (nanovtt::ZeroCrossing(iRay, acc, ijk, v, t0)) {
                // write distance to surface. (we assume it is a uniform voxel)
                float wT0 = t0 * float(grid->voxelSize()[0]);
                compositeOp(image, i, width, height, wT0 / (wBBoxDimZ * 2), 1.0f);
            }
            else {
                // write background value.
                compositeOp(image, i, width, height, 0.0f, 0.0f);
            }
        }
    };

    // Original HDDA approach for PNanoVTT
    auto renderOpPNanoVTT = [width, height, rayGenOp, compositeOp, treeIndexBbox, wBBoxDimZ] (int start, int end, float* image, const GridT * grid) {

        const pnanovdb_grid_type_t grid_type = PNANOVDB_GRID_TYPE_FLOAT;
        const pnanovdb_buf_t buf = pnanovdb_make_buf(const_cast<uint32_t*>(
            reinterpret_cast<uint32_t const*>(grid->tree().getContainer())),
            grid->tree().getContainer()->m_bufferSize);
        const pnanovdb_grid_handle_t pGridHandle{ pnanovdb_address_offset64(pnanovdb_address_null(), grid->tree().getContainer()->m_containerSize) };
        const pnanovtt_tree_handle_t pTreeHandle{ pnanovdb_address_offset64(pnanovdb_address_null(), grid->tree().getContainer()->m_containerSize + PNANOVDB_GRID_SIZE) };

        pnanovtt_readaccessor_t pacc;
        pnanovtt_readaccessor_init(buf, PNANOVDB_REF(pacc), pGridHandle, pTreeHandle);

        for (int i = start; i < end; ++i) {
            Vec3T rayEye;
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);
            // generate ray.
            RayT wRay(rayEye, rayDir);
            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);
            // intersect...
            float  t0 = 0.f;
            float vHit = 0.f;
            pnanovdb_coord_t ijk;
            const pnanovdb_vec3_t origin{ iRay.eye()[0], iRay.eye()[1], iRay.eye()[2] };
            const pnanovdb_vec3_t direction{ iRay.dir()[0], iRay.dir()[1], iRay.dir()[2] };
            const float tMin = iRay.t0();
            const float tMax = iRay.t1();

            const pnanovdb_bool_t foundZC = pnanovtt_hdda_zero_crossing(
                grid_type,
                buf,
                PNANOVDB_REF(pacc),
                PNANOVDB_REF(origin),
                tMin,
                PNANOVDB_REF(direction),
                tMax,
                PNANOVDB_REF(t0),
                PNANOVDB_REF(vHit),
                PNANOVDB_REF(ijk),
                PNANOVTT_TRAITS_MAX_DEPTH);
            if (foundZC) {
                // write distance to surface. (we assume it is a uniform voxel)
                float wT0 = t0 * float(grid->voxelSize()[0]);
                compositeOp(image, i, width, height, wT0 / (wBBoxDimZ * 2), 1.0f);
            }
            else {
                // write background value.
                compositeOp(image, i, width, height, 0.0f, 0.0f);
            }
        }
    };

    if (numCPUIterations > 0)
    {
        float durationAvg = 0;
        for (int i = 0; i < numCPUIterations; ++i) {
            float duration = 0;
            if (useSubVoxelAccuracy)
            {
                duration = renderImage(false, renderOpSubVoxel, width, height, h_outImage, h_grid);
            }
            else {
                duration = renderImage(false, renderOp, width, height, h_outImage, h_grid);
            }
            //std::cout << "Duration(NanoVTT-Host) = " << duration << " ms" << std::endl;
            durationAvg += duration;
        }
        durationAvg /= numCPUIterations;
        std::cout << "Average Duration(NanoVTT-Host) = " << durationAvg << " ms" << std::endl;
        saveImage(std::string("raytrace_level_set") + imagePostFix + "-nanovtt-host.pfm", width, height, (float*)imageBuffer.data());
    }

    if (numCPUIterations > 0)
    {
        float durationAvg = 0;
        for (int i = 0; i < numCPUIterations; ++i) {
            float duration = 0;
            if (useSubVoxelAccuracy)
            {
                duration = renderImageHost(renderOpSubVoxelPNanoVTT, width, height, h_outImage, h_grid);
            }
            else {
                duration = renderImageHost(renderOpPNanoVTT, width, height, h_outImage, h_grid);
            }
            //std::cout << "Duration(PNanoVTT-Host) = " << duration << " ms" << std::endl;
            durationAvg += duration;
        }
        durationAvg /= numCPUIterations;
        std::cout << "Average Duration(PNanoVTT-Host) = " << durationAvg << " ms" << std::endl;
        saveImage(std::string("raytrace_level_set") + imagePostFix + "-pnanovtt-host.pfm", width, height, (float*)imageBuffer.data());
    }

#if defined(NANOVDB_USE_CUDA)
    if (numGPUIterations > 0)
    {
        handle.deviceUpload();

        auto* d_grid = handle.template deviceGrid<float>();
        if (!d_grid)
            throw std::runtime_error("GridHandle does not contain a valid device grid");

        imageBuffer.deviceUpload();
        float* d_outImage = reinterpret_cast<float*>(imageBuffer.deviceData());

        {
            float durationAvg = 0;
            for (int i = 0; i < numGPUIterations; ++i) {
                float duration = 0;
                if (useSubVoxelAccuracy)
                {
                    duration = renderImage(true, renderOpSubVoxel, width, height, d_outImage, d_grid);
                }
                else {
                    duration = renderImage(true, renderOp, width, height, d_outImage, d_grid);
                }
                //std::cout << "Duration(NanoVTT-Cuda) = " << duration << " ms" << std::endl;
                durationAvg += duration;
            }
            durationAvg /= numGPUIterations;
            std::cout << "Average Duration(NanoVTT-Cuda) = " << durationAvg << " ms" << std::endl;
            imageBuffer.deviceDownload();
            saveImage(std::string("raytrace_level_set") + imagePostFix + "-nanovtt-cuda.pfm", width, height, (float*)imageBuffer.data());
        }
    }
#endif
}
